#include "hip/hip_runtime.h"
#include "Scene.hpp"
#include "Sphere.hpp"
#include "Triangle.hpp"

using namespace std;

Scene::Scene() {
	lights.clear();
	triangles.clear();
	spheres.clear();
}

Scene::~Scene() {
	lights.clear();
	triangles.clear();
	spheres.clear();
}

void Scene::setupCudaMem(int bufferSize) {
   /* allocate device memory */
   checkCudaErrors(hipMalloc((void**) pixels_d, bufferSize), "hipMalloc");
   checkCudaErrors(hipMalloc((void**) lights_d, lights.size()), "hipMalloc");
   checkCudaErrors(hipMalloc((void**) triangles_d, triangles.size()), "hipMalloc");
   checkCudaErrors(hipMalloc((void**) spheres_d, spheres.size()), "hipMalloc");
   checkCudaErrors(hipMalloc((void**) planes_d, planes.size()), "hipMalloc");

   /* copy into device memory */
   checkCudaErrors(hipMemcpy((void **) &lights_d, &(lights[0]), lights.size(),
                   hipMemcpyHostToDevice), "hipMemcpy");
   checkCudaErrors(hipMemcpy((void **) triangles_d, &(triangles[0]), triangles.size(),
                   hipMemcpyHostToDevice), "hipMemcpy");
   checkCudaErrors(hipMemcpy((void **) spheres_d, &(spheres[0]), spheres.size(),
                   hipMemcpyHostToDevice), "hipMemcpy");
   checkCudaErrors(hipMemcpy((void **) planes_d, &(planes[0]), planes.size(),
                   hipMemcpyHostToDevice), "hipMemcpy");
}

void Scene::getCudaMem(Pixel *pixels_h, int bufferSize) {
   checkCudaErrors(hipMemcpy(pixels_h, pixels_d, bufferSize,
                   hipMemcpyDeviceToHost), "hipMemcpy");
   //TODO memcopy back for pixels is correct???
   //TODO free gpu mem
   
}

void renderStart(int width, int height,
                 Vector3f backgroundCol, Vector3f CameraRight,
                 Vector3f CameraUp, Vector3f CameraPos,
                 Vector3f CameraDirection, Pixel *pixels,
                 Light *lights, int numLights,
                 Plane *planes, int numPlanes,
                 Triangle *triangles, int numTriangles,
                 Sphere *spheres, int numSpheres)
{
   float aspectRatio = (float) width / height;
   dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
   dim3 dimGrid(ceil((double) width / TILE_WIDTH),
                ceil((double) height / TILE_WIDTH));

   renderScene<<<dimGrid, dimBlock>>>(aspectRatio, 
                                      backgroundCol, CameraRight,
                                      CameraUp, CameraPos,
                                      CameraDirection, pixels,
                                      lights, numLights,
                                      planes, numPlanes,
                                      triangles, numTriangles,
                                      spheres, numSpheres);
}

/*
 * Check for errors on cuda calls and exit if one is thrown.
 */
void checkCudaErrors(int errorCode, char const *callName) {
   if(errorCode > 0) {
      printf("Cuda Error: %s\n", callName);
      exit(EXIT_FAILURE);
   }
}

__device__ hit_t checkHit(Ray testRay, Shape *exclude,
                          Plane *planes, int numPlanes,
                          Triangle *triangles, int numTriangles,
                          Sphere *spheres, int numSpheres)
{
	Shape* hitShape = NULL;
	bool hit = false;
	float bestT = 10000;

	for (unsigned int i = 0; i < numPlanes; ++i)
	{
      if(&(planes[i]) != exclude) {
		   float t = planes[i].checkHit(testRay.eye, testRay.direction);
		   if (t > 0 && t < bestT) {
			   hitShape = &(planes[i]);
			   bestT = t;
   			hit = true;
         }
		}
	}

	for (unsigned int i = 0; i < numTriangles; ++i)
	{
      if(&(triangles[i]) != exclude) {
   		float t = triangles[i].checkHit(testRay.eye, testRay.direction);
	   	if (t > 0 && t < bestT) {
		   	hitShape = &(triangles[i]);
			   bestT = t;
   			hit = true;
         }
		}
	}

	for (unsigned int i = 0; i < numSpheres; ++i)
	{
      if(&(spheres[i]) != exclude) {
   		float t = spheres[i].checkHit(testRay.eye, testRay.direction);
	   	if (t > 0 && t < bestT) {
		   	hitShape = &(spheres[i]);
			   bestT = t;
   			hit = true;
         }
		}
	}

	if (!hit) {
		hitShape = NULL;
	}

	hit_t ret;
	ret.hitShape = hitShape;
	ret.isHit = hit;
	ret.t = bestT;

	return ret;
}

__device__ Pixel ComputeLighting(Ray laser, hit_t hitResult,
                                 Light *lights, int numLights,
                                 Plane *planes, int numPlanes,
                                 Triangle *triangles, int numTriangles,
                                 Sphere *spheres, int numSpheres)
{
	Vector3f hitPt = laser.eye + laser.direction*hitResult.t;
	Vector3f viewVec = -laser.direction;
	Vector3f rgb = hitResult.hitShape->mat.rgb;
	Vector3f ambient = rgb*hitResult.hitShape->mat.ambient;
   Vector3f n = hitResult.hitShape->GetNormal(hitPt);
	Vector3f color;
	bool inShadow;

	// calculate if the point is in a shadow. If so, we later return the pixel as all black
	for (int i = 0; i < numLights; ++i)
	{
		inShadow = false;
		Vector3f shadowDir = normalize(lights[i].location - hitPt);
	   Vector3f l = shadowDir;//normalize(lights[i].location - hitPt);
		Ray shadowRay = Ray(hitPt, shadowDir);
		hit_t shadowHit = checkHit(shadowRay, hitResult.hitShape,
                                 planes, numPlanes,
                                 triangles, numTriangles,
                                 spheres, numSpheres);

		if (shadowHit.isHit) {
			if (shadowHit.hitShape != hitResult.hitShape)
				inShadow = true;
		}

      if (!inShadow) {
         Vector3f r = -l + n * 2 * dot(n,l);
         r = normalize(r);

         float specMult = max(dot(viewVec, r), 0.0f);
         specMult = pow(specMult, hitResult.hitShape->mat.shine);
         
         Vector3f colorS = rgb * specMult;

			float hold = min(max(dot(l, n), 0.0f), 1.0f);
			Vector3f colorD = rgb * hold;

			Vector3f toAdd = colorD * hitResult.hitShape->mat.diffuse
                               + colorS * hitResult.hitShape->mat.specular;
         //spec + diffuse setup
			toAdd[0] *= lights[i].color.r;
			toAdd[1] *= lights[i].color.g;
			toAdd[2] *= lights[i].color.b;
         //actually add spec + diffuse
			color = color + toAdd;
		}
      //ambient addition
	   color[0] += ambient[0] * lights[i].color.r;
	   color[1] += ambient[1] * lights[i].color.g;
	   color[2] += ambient[2] * lights[i].color.b;
      //make sure in range still
      color[0] = min(max(color[0],0.0f),1.0f);
      color[1] = min(max(color[1],0.0f),1.0f);
      color[2] = min(max(color[2],0.0f),1.0f);
	}

	return Pixel(color[0], color[1], color[2]);
}

__global__ void renderScene(float aspectRatio,
                            Vector3f backgroundCol, Vector3f CameraRight,
                            Vector3f CameraUp, Vector3f CameraPos,
                            Vector3f CameraDirection, Pixel *pixels,
                            Light *lights, int numLights,
                            Plane *planes, int numPlanes,
                            Triangle *triangles, int numTriangles,
                            Sphere *spheres, int numSpheres)
{
   int row = blockIdx.y * blockDim.y + threadIdx.y;
   int col = blockIdx.x * blockDim.x + threadIdx.x;

   float normalized_i, normalized_j;
   if(aspectRatio > 1) {
      normalized_i = ((col/(float)gridDim.x) - 0.5) * aspectRatio;
      normalized_j = (row/(float)gridDim.y) - 0.5;
   }
   else {
      normalized_i = (col/(float)gridDim.x) - 0.5;
      normalized_j = ((row/(float)gridDim.y) - 0.5) / aspectRatio;
   }

   Vector3f imagePoint = CameraRight * normalized_i + 
                         CameraUp * normalized_j +
                         CameraPos + CameraDirection;

   Vector3f ray_direction = imagePoint - CameraPos;

   Ray laser = Ray(CameraPos, ray_direction);

   // no shape to exclude so use NULL 
   hit_t hitShape = checkHit(laser, NULL,
                             planes, numPlanes,
                             triangles, numTriangles,
                             spheres, numSpheres);
   
   if (hitShape.isHit) {
      pixels[col + row * gridDim.x] = ComputeLighting(laser, hitShape,
                                                  lights, numLights,
                                                  planes, numPlanes,
                                                  triangles, numTriangles,
                                                  spheres, numSpheres);
   } else {
      // not hit means we color it with a background color
      pixels[col + row * gridDim.x] = Pixel(backgroundCol[0],
                                            backgroundCol[1],
                                            backgroundCol[2]);
   }
}

